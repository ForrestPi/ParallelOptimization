#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"    
#include <iostream>  
#include <stdio.h>    
#include <math.h>    
 
#define N (1024*1024)    
#define FULL_DATA_SIZE N*20    
 
__global__ void kernel(int* a, int *b, int*c)
{
	int threadID = blockIdx.x * blockDim.x + threadIdx.x;
 
	if (threadID < N)
	{
		c[threadID] = (a[threadID] + b[threadID]) / 2;
	}
}
 
int main()
{
	//获取设备属性  
	hipDeviceProp_t prop;
	int deviceID;
	hipGetDevice(&deviceID);
	hipGetDeviceProperties(&prop, deviceID);
 
	//检查设备是否支持重叠功能  
	if (!prop.deviceOverlap)
	{
		printf("No device will handle overlaps. so no speed up from stream.\n");
		return 0;
	}
 
	//启动计时器  
	hipEvent_t start, stop;
	float elapsedTime;
	hipEventCreate(&start);
	hipEventCreate(&stop);
	hipEventRecord(start, 0);
 
	//创建两个CUDA流  
	hipStream_t stream, stream1;
	hipStreamCreate(&stream);
	hipStreamCreate(&stream1);
 
	int *host_a, *host_b, *host_c;
	int *dev_a, *dev_b, *dev_c;
	int *dev_a1, *dev_b1, *dev_c1;
 
	//在GPU上分配内存  
	hipMalloc((void**)&dev_a, N * sizeof(int));
	hipMalloc((void**)&dev_b, N * sizeof(int));
	hipMalloc((void**)&dev_c, N * sizeof(int));
 
	hipMalloc((void**)&dev_a1, N * sizeof(int));
	hipMalloc((void**)&dev_b1, N * sizeof(int));
	hipMalloc((void**)&dev_c1, N * sizeof(int));
 
	//在CPU上分配页锁定内存  
	hipHostAlloc((void**)&host_a, FULL_DATA_SIZE * sizeof(int), hipHostMallocDefault);
	hipHostAlloc((void**)&host_b, FULL_DATA_SIZE * sizeof(int), hipHostMallocDefault);
	hipHostAlloc((void**)&host_c, FULL_DATA_SIZE * sizeof(int), hipHostMallocDefault);
 
	//主机上的内存赋值  
	for (int i = 0; i < FULL_DATA_SIZE; i++)
	{
		host_a[i] = i;
		host_b[i] = FULL_DATA_SIZE - i;
	}
 
	for (int i = 0; i < FULL_DATA_SIZE; i += 2 * N)
	{
		hipMemcpyAsync(dev_a, host_a + i, N * sizeof(int), hipMemcpyHostToDevice, stream);
		hipMemcpyAsync(dev_b, host_b + i, N * sizeof(int), hipMemcpyHostToDevice, stream);
 
		hipMemcpyAsync(dev_a1, host_a + i + N, N * sizeof(int), hipMemcpyHostToDevice, stream1);
		hipMemcpyAsync(dev_b1, host_b + i + N, N * sizeof(int), hipMemcpyHostToDevice, stream1);
 
		kernel << <N / 1024, 1024, 0, stream >> > (dev_a, dev_b, dev_c);
		kernel << <N / 1024, 1024, 0, stream1 >> > (dev_a, dev_b, dev_c1);
 
		hipMemcpyAsync(host_c + i, dev_c, N * sizeof(int), hipMemcpyDeviceToHost, stream);
		hipMemcpyAsync(host_c + i + N, dev_c1, N * sizeof(int), hipMemcpyDeviceToHost, stream1);
	}
 
	// 等待Stream流执行完成
	hipStreamSynchronize(stream);
	hipStreamSynchronize(stream1);
 
	hipEventRecord(stop, 0);
	hipEventSynchronize(stop);
	hipEventElapsedTime(&elapsedTime, start, stop);
 
	std::cout << "消耗时间： " << elapsedTime << std::endl;
 
	//输出前10个结果  
	for (int i = 0; i < 10; i++)
	{
		std::cout << host_c[i] << std::endl;
	}
 
	getchar();
 
	// free stream and mem    
	hipHostFree(host_a);
	hipHostFree(host_b);
	hipHostFree(host_c);
 
	hipFree(dev_a);
	hipFree(dev_b);
	hipFree(dev_c);
 
	hipFree(dev_a1);
	hipFree(dev_b1);
	hipFree(dev_c1);
 
	hipStreamDestroy(stream);
	hipStreamDestroy(stream1);
	return 0;
}

//使用2个流，执行时间16ms，基本上是使用一个流消耗时间的二分之一。